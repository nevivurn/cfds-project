#include "hip/hip_runtime.h"
#include <math.h>
#include <mpi.h>
#include <cassert>

#include "classifier.h"
#include "util.h"

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

static int mpi_rank;

// Multi-dimensional matrix containing fp32 elements
struct Tensor {
  Tensor(std::vector<int> shape_);
  Tensor(std::vector<int> shape_, float *buf_);
  ~Tensor();
  __host__ __device__ int num_elem() const;
  void fill_zeros();

  void allocate_gpu();
  void copy_to_gpu(hipStream_t stream = 0);
  void copy_to_cpu(hipStream_t stream = 0);

  float *buf = nullptr;
  float *gbuf = nullptr;
  int ndim = 0;
  int shape[4];
  int datashape[4];
};

Tensor::Tensor(std::vector<int> shape_) {
  ndim = shape_.size();
  for (int i = 0; i < ndim; ++i) { shape[i] = datashape[i] = shape_[i]; }
  int N_ = num_elem();
  buf = (float *) calloc(N_, sizeof(float));
}

Tensor::Tensor(std::vector<int> shape_, float *buf_) {
  ndim = shape_.size();
  for (int i = 0; i < ndim; ++i) { shape[i] = datashape[i] = shape_[i]; }
  int N_ = num_elem();
  buf = (float *) calloc(N_, sizeof(float));
  for (int n = 0; n < N_; ++n) { buf[n] = buf_[n]; }
}

Tensor::~Tensor() {
  // TODO(nevi): properly free memory
  //if (buf != nullptr) free(buf);
  //if (gbuf != nullptr) hipFree(gbuf);
}

int Tensor::num_elem() const {
  int sz = 1;
  for (int i = 0; i < ndim; ++i) { sz *= shape[i]; }
  return sz;
}

void Tensor::fill_zeros() {
  int N_ = num_elem();
  for (int n = 0; n < N_; ++n) { buf[n] = 0.0; }
}

void Tensor::allocate_gpu() {
  CHECK_CUDA(hipMalloc(&gbuf, num_elem() * sizeof(float)));
}

void Tensor::copy_to_gpu(hipStream_t stream) {
  if (gbuf == nullptr)
    allocate_gpu();
  CHECK_CUDA(hipMemcpyAsync(gbuf, buf, num_elem() * sizeof(float), hipMemcpyHostToDevice, stream));
}

void Tensor::copy_to_cpu(hipStream_t stream) {
  CHECK_CUDA(hipMemcpyAsync(buf, gbuf, num_elem() * sizeof(float), hipMemcpyDeviceToHost, stream));
}

// Parameters
Tensor *w_conv1, *w_conv2, *w_conv3, *w_conv4, *w_conv5, *w_conv6, *b_conv1,
    *b_conv2, *b_conv3, *b_conv4, *b_conv5, *b_conv6, *w_fc1, *w_fc2, *w_fc3,
    *b_fc1, *b_fc2, *b_fc3, *gamma_conv1, *beta_conv1, *gamma_conv6, *beta_conv6;

// Activations
Tensor *a_conv1, *a_layernorm1, *a_relu1, *a_pool1;
Tensor *a_conv1_sum, *a_conv1_sum_sq; // for layernorm
Tensor *a_conv2, *a_relu2, *a_pool2;
Tensor *a_conv3, *a_relu3;
Tensor *a_conv4, *a_relu4;
Tensor *a_conv5, *a_relu5;
Tensor *a_conv6, *a_layernorm6, *a_relu6, *a_pool6;
Tensor *a_conv6_sum, *a_conv6_sum_sq; // for layernorm
Tensor *a_collapse;
Tensor *a_linear1, *a_relu7;
Tensor *a_linear2, *a_relu8;
Tensor *a_linear3;

// Operations
void conv1d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride, int padding, int dilation, bool has_bias);
void relu(Tensor *input, Tensor *output);
void maxpool1d(Tensor *input, Tensor *output, int kernel_size, int stride);
void collapse(Tensor *input, Tensor *output);
void linear(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            bool has_bias);
void layernorm(Tensor *input, Tensor *gamma, Tensor *beta, Tensor *output);

// Cuda layers
__global__ void cuda_conv1d(Tensor input, Tensor weight, Tensor bias, Tensor output);
__global__ void cuda_relu(Tensor input, Tensor output);
__global__ void cuda_maxpool1d(Tensor input, Tensor output); // always 3x3
__global__ void cuda_collapse(Tensor input, Tensor output);
__global__ void cuda_linear(Tensor input, Tensor weight, Tensor bias, Tensor output);
__global__ void cuda_layernorm(Tensor input, Tensor gamma, Tensor beta, Tensor output,
                               Tensor sum, Tensor sum_sq);

// Cuda operations
__global__ void cuda_reduce_sum(Tensor input, Tensor output);
__global__ void cuda_reduce_sum_sq(Tensor input, Tensor output);

// Parallelization method is totally up to you, but you should gather
// the output at rank 0
void classifier(float *input_, float *output_, int N) {
  if (mpi_rank == 0) {
    for (int n = 0; n < N; ++n) {  // N input sentences

      // Load one input sentence from input
      Tensor *one_input = new Tensor({1, VOCAB_SIZE, MAX_LENGTH}, input_ + n * VOCAB_SIZE * MAX_LENGTH);

      // Conv block 1 : Conv1d + LayerNorm + ReLU + MaxPool1d
      one_input->copy_to_gpu();
      cuda_conv1d<<<256,1008>>>(*one_input, *w_conv1, *b_conv1, *a_conv1);
      cuda_reduce_sum<<<1008, 256, 256 * sizeof(float)>>>(*a_conv1, *a_conv1_sum);
      cuda_reduce_sum<<<1, 1024, 1024 * sizeof(float)>>>(*a_conv1_sum, *a_conv1_sum);
      cuda_reduce_sum_sq<<<1008, 256, 256 * sizeof(float)>>>(*a_conv1, *a_conv1_sum_sq);
      cuda_reduce_sum<<<1, 1024, 1024 * sizeof(float)>>>(*a_conv1_sum_sq, *a_conv1_sum_sq);
      cuda_layernorm<<<256, 1008>>>(*a_conv1, *gamma_conv1, *beta_conv1, *a_layernorm1,
                                    *a_conv1_sum, *a_conv1_sum_sq);
      cuda_relu<<<256, 1008>>>(*a_layernorm1, *a_relu1);
      cuda_maxpool1d<<<256, 336>>>(*a_relu1, *a_pool1);

      // Conv block 2 : Conv1d + ReLU + MaxPool1d
      cuda_conv1d<<<256, 330>>>(*a_pool1, *w_conv2, *b_conv2, *a_conv2);
      cuda_relu<<<256, 330>>>(*a_conv2, *a_relu2);
      cuda_maxpool1d<<<256, 110>>>(*a_relu2, *a_pool2);

      // Conv block 3 : Conv1d + ReLU
      cuda_conv1d<<<256, 108>>>(*a_pool2, *w_conv3, *b_conv3, *a_conv3);
      cuda_relu<<<256, 108>>>(*a_conv3, *a_relu3);

      // Conv block 4 : Conv1d + ReLU
      cuda_conv1d<<<256, 106>>>(*a_relu3, *w_conv4, *b_conv4, *a_conv4);
      cuda_relu<<<256, 106>>>(*a_conv4, *a_relu4);

      // Conv block 5 : Conv1d + ReLU
      cuda_conv1d<<<256, 104>>>(*a_relu4, *w_conv5, *b_conv5, *a_conv5);
      cuda_relu<<<256, 104>>>(*a_conv5, *a_relu5);

      // Conv block 6 : Conv1d + LayerNorm + ReLU + MaxPool1d
      cuda_conv1d<<<256, 102>>>(*a_relu5, *w_conv6, *b_conv6, *a_conv6);
      cuda_reduce_sum<<<102, 256, 256 * sizeof(float)>>>(*a_conv6, *a_conv6_sum);
      cuda_reduce_sum<<<1, 128, 128 * sizeof(float)>>>(*a_conv6_sum, *a_conv6_sum);
      cuda_reduce_sum_sq<<<102, 256, 256 * sizeof(float)>>>(*a_conv6, *a_conv6_sum_sq);
      cuda_reduce_sum<<<1, 128, 128 * sizeof(float)>>>(*a_conv6_sum_sq, *a_conv6_sum_sq);
      cuda_layernorm<<<256, 102>>>(*a_conv6, *gamma_conv6, *beta_conv6, *a_layernorm6,
                                    *a_conv6_sum, *a_conv6_sum_sq);
      cuda_relu<<<256, 102>>>(*a_layernorm6, *a_relu6);
      cuda_maxpool1d<<<256, 34>>>(*a_relu6, *a_pool6);

      // Collapse
      cuda_collapse<<<68, 128>>>(*a_pool6, *a_collapse);

      // FC block 1 : Linear + ReLU
      cuda_linear<<<1, 1024>>>(*a_collapse, *w_fc1, *b_fc1, *a_linear1);
      cuda_relu<<<1, 1024>>>(*a_linear1, *a_relu7);

      // FC block 2 : Linear + ReLU
      cuda_linear<<<1, 1024>>>(*a_relu7, *w_fc2, *b_fc2, *a_linear2);
      cuda_relu<<<1, 1024>>>(*a_linear2, *a_relu8);

      // FC block 3 : Linear
      cuda_linear<<<1, 4>>>(*a_relu8, *w_fc3, *b_fc3, *a_linear3);

      a_linear3->copy_to_cpu();
      CHECK_CUDA(hipStreamSynchronize(0));

      float max_val = -1e99f;
      int max_idx = 0;
      for (int i = 0; i < a_linear3->num_elem(); ++i) {
        if (a_linear3->buf[i] > max_val) {
          max_val = a_linear3->buf[i];
          max_idx = i;
        }
      }

      output_[n] = max_idx;
    }  // end N input sentences loop
  }    // if mpi_rank == 0
}

__global__ void cuda_conv1d(Tensor input, Tensor weight, Tensor bias, Tensor output) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int in_channels = weight.shape[1];
  int kernel_size = weight.shape[2];
  int input_length = input.shape[2];
  int output_length = output.shape[2];

  int oc = i / output_length;
  int ol = i % output_length;

  float val = 0.0f;
  for (int ic = 0; ic < in_channels; ++ic)
    for (int ks = 0; ks < kernel_size; ++ks)
      val += weight.gbuf[oc * in_channels * kernel_size + ic * kernel_size + ks] *
        input.gbuf[ic * input_length + ks + ol];
  val += bias.gbuf[oc];
  output.gbuf[i] = val;
}

void conv1d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride = 1, int padding = 0, int dilation = 1,
            bool has_bias = true) {
  int out_channels = weight->shape[0];
  int in_channels = weight->shape[1];
  int kernel_size = weight->shape[2];
  int input_length = input->shape[2];
  int output_length =
      (input->shape[2] + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

  for (int oc = 0; oc < out_channels; ++oc) {
    for (int ol = 0; ol < output_length; ++ol) {
      float val = 0.0f;
      int offset = ol;
      for (int ic = 0; ic < in_channels; ++ic) {
        for (int ks = 0; ks < kernel_size; ++ks) {
          val += weight->buf[oc * in_channels * kernel_size + ic * kernel_size + ks] *
                 input->buf[ic * input_length + ks + offset];
        }
      }
      if (has_bias) val += bias->buf[oc];
      output->buf[oc * output_length + ol] = val;
    }
  }
}


__global__ void cuda_relu(Tensor input, Tensor output) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (input.gbuf[i] > 0.0f)
    output.gbuf[i] = input.gbuf[i];
  else
    output.gbuf[i] = 0.0f;
}

void relu(Tensor *input, Tensor *output) {
  for (int i = 0; i < input->num_elem(); ++i) {
    if (input->buf[i] > 0.0f)
      output->buf[i] = input->buf[i];
    else
      output->buf[i] = 0.0f;
  }
}

__global__ void cuda_maxpool1d(Tensor input, Tensor output) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int input_length = input.shape[2];
  int output_length = output.shape[2];

  int oc = i / output_length;
  int ol = i % output_length;

  float mx = -1e99f;
  for (int ks = 0; ks < 3; ++ks) {
    float val = input.gbuf[oc * input_length + ks + ol * 3];
    if (val > mx) mx = val;
  }
  output.gbuf[i] = mx;
}

void maxpool1d(Tensor *input, Tensor *output, int kernel_size, int stride) {
  int IL = input->shape[2];
  int OC = output->shape[1];
  int OL = output->shape[2];

  for (int oc = 0; oc < OC; ++oc) {
    for (int ol = 0; ol < OL; ++ol) {
      float mx = -1e99;
      for (int ks = 0; ks < kernel_size; ++ks) {
        float val = input->buf[oc * IL + ks + ol * stride];
        if (val > mx) mx = val;
      }
      output->buf[oc * OL + ol] = mx;
    }
  }
}

__global__ void cuda_collapse(Tensor input, Tensor output) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  output.gbuf[i] = input.gbuf[i];
}

void collapse(Tensor *input, Tensor *output) {
  for (int i = 0; i < input->num_elem(); ++i) {
    output->buf[i] = input->buf[i];
  }
}

__global__ void cuda_linear(Tensor input, Tensor weight, Tensor bias, Tensor output) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int IC = input.shape[1];
  int OC = output.shape[1];

  float val = bias.gbuf[i];
  for (int ic = 0; ic < IC; ++ic)
    val += input.gbuf[ic] * weight.gbuf[i * IC + ic];
  output.gbuf[i] = val;
}

void linear(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            bool has_bias) {
  int IC = input->shape[1];
  int OC = output->shape[1];

  for (int oc = 0; oc < OC; ++oc) {
    float val = 0.0;
    for (int ic = 0; ic < IC; ++ic) {
      val += input->buf[ic] * weight->buf[oc * IC + ic];
    }
    if (has_bias) val += bias->buf[oc];
    output->buf[oc] = val;
  }
}

__global__ void cuda_reduce_sum(Tensor input, Tensor output) {
  extern __shared__ float sdata[];
  int tid = threadIdx.x;
  sdata[tid] = input.gbuf[blockIdx.x * blockDim.x + tid];
  __syncthreads();
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    output.gbuf[blockIdx.x] = sdata[0];
}

__global__ void cuda_reduce_sum_sq(Tensor input, Tensor output) {
  extern __shared__ float sdata[];
  int tid = threadIdx.x;
  sdata[tid] = input.gbuf[blockIdx.x * blockDim.x + tid];
  sdata[tid] *= sdata[tid];
  __syncthreads();
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    output.gbuf[blockIdx.x] = sdata[0];
}

__global__ void cuda_layernorm(Tensor input, Tensor gamma, Tensor beta, Tensor output,
                               Tensor sum, Tensor sum_sq) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // E[X], E[X^2]
  float mean1 = sum.gbuf[0] / input.num_elem();
  float mean2 = sum_sq.gbuf[0] / (float)input.num_elem();

  // V[X]
  float var = mean2 - mean1 * mean1;

  output.gbuf[idx] = (input.gbuf[idx] - mean1) / sqrtf(var + 1e-5) * gamma.gbuf[idx] + beta.gbuf[idx];
}

void layernorm(Tensor *input, Tensor *gamma, Tensor *beta, Tensor *output) {
  // E[X], E[X^2]
  float sum1 = 0.0f, sum2 = 0.0f;
  for (int i = 0; i < input->num_elem(); ++i) {
      sum1 += input->buf[i];
      sum2 += input->buf[i] * input->buf[i];
  }
  float mean1 = sum1 / (float)input->num_elem();
  float mean2 = sum2 / (float)input->num_elem();

  // V[X]
  float var = mean2 - mean1 * mean1;

  // Normalization
  for (int i = 0; i < input->num_elem(); ++i) {
    output->buf[i] = (input->buf[i] - mean1) / sqrtf(var + 1e-5) * gamma->buf[i] + beta->buf[i];
  }
}

// load the parameter binary file and store parameters into Tensors
// Only the first process (root, mpi_rank == 0) has the parameter
// You must broadcast it to the others
void initialize_classifier(float *parameter, int N) {
  CHECK_CUDA(hipSetDevice(0));

  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  if (mpi_rank == 0) {
    w_conv1 = new Tensor({256, 70, 7}, parameter + OFFSET0);
    w_conv1->copy_to_gpu();
    b_conv1 = new Tensor({256}, parameter + OFFSET1);
    b_conv1->copy_to_gpu();
    gamma_conv1 = new Tensor({256, 1008}, parameter + OFFSET2);
    gamma_conv1->copy_to_gpu();
    beta_conv1 = new Tensor({256, 1008}, parameter + OFFSET3);
    beta_conv1->copy_to_gpu();

    w_conv2 = new Tensor({256, 256, 7}, parameter + OFFSET4);
    w_conv2->copy_to_gpu();
    b_conv2 = new Tensor({256}, parameter + OFFSET5);
    b_conv2->copy_to_gpu();
    w_conv3 = new Tensor({256, 256, 3}, parameter + OFFSET6);
    w_conv3->copy_to_gpu();
    b_conv3 = new Tensor({256}, parameter + OFFSET7);
    b_conv3->copy_to_gpu();
    w_conv4 = new Tensor({256, 256, 3}, parameter + OFFSET8);
    w_conv4->copy_to_gpu();
    b_conv4 = new Tensor({256}, parameter + OFFSET9);
    b_conv4->copy_to_gpu();
    w_conv5 = new Tensor({256, 256, 3}, parameter + OFFSET10);
    w_conv5->copy_to_gpu();
    b_conv5 = new Tensor({256}, parameter + OFFSET11);
    b_conv5->copy_to_gpu();

    w_conv6 = new Tensor({256, 256, 3}, parameter + OFFSET12);
    w_conv6->copy_to_gpu();
    b_conv6 = new Tensor({256}, parameter + OFFSET13);
    b_conv6->copy_to_gpu();
    gamma_conv6 = new Tensor({256, 102}, parameter + OFFSET14);
    gamma_conv6->copy_to_gpu();
    beta_conv6 = new Tensor({256, 102}, parameter + OFFSET15);
    beta_conv6->copy_to_gpu();
    w_fc1 = new Tensor({1024, 8704}, parameter + OFFSET16);
    w_fc1->copy_to_gpu();
    b_fc1 = new Tensor({1024}, parameter + OFFSET17);
    b_fc1->copy_to_gpu();
    w_fc2 = new Tensor({1024, 1024}, parameter + OFFSET18);
    w_fc2->copy_to_gpu();
    b_fc2 = new Tensor({1024}, parameter + OFFSET19);
    b_fc2->copy_to_gpu();
    w_fc3 = new Tensor({4, 1024}, parameter + OFFSET20);
    w_fc3->copy_to_gpu();
    b_fc3 = new Tensor({4}, parameter + OFFSET21);
    b_fc3->copy_to_gpu();

    a_conv1 = new Tensor({1, 256, 1008});
    a_conv1->allocate_gpu();
    a_conv1_sum = new Tensor({1, 1024}); // rounded up
    a_conv1_sum->allocate_gpu();
    a_conv1_sum_sq = new Tensor({1, 1024});
    a_conv1_sum_sq->allocate_gpu();
    a_layernorm1 = new Tensor({1, 256, 1008});
    a_layernorm1->allocate_gpu();
    a_relu1 = new Tensor({1, 256, 1008});
    a_relu1->allocate_gpu();
    a_pool1 = new Tensor({1, 256, 336});
    a_pool1->allocate_gpu();

    a_conv2 = new Tensor({1, 256, 330});
    a_conv2->allocate_gpu();
    a_relu2 = new Tensor({1, 256, 330});
    a_relu2->allocate_gpu();
    a_pool2 = new Tensor({1, 256, 110});
    a_pool2->allocate_gpu();

    a_conv3 = new Tensor({1, 256, 108});
    a_conv3->allocate_gpu();
    a_relu3 = new Tensor({1, 256, 108});
    a_relu3->allocate_gpu();

    a_conv4 = new Tensor({1, 256, 106});
    a_conv4->allocate_gpu();
    a_relu4 = new Tensor({1, 256, 106});
    a_relu4->allocate_gpu();

    a_conv5 = new Tensor({1, 256, 104});
    a_conv5->allocate_gpu();
    a_relu5 = new Tensor({1, 256, 104});
    a_relu5->allocate_gpu();

    a_conv6 = new Tensor({1, 256, 102});
    a_conv6->allocate_gpu();
    a_conv6_sum = new Tensor({1, 128}); // rounded up
    a_conv6_sum->allocate_gpu();
    a_conv6_sum_sq = new Tensor({1, 128});
    a_conv6_sum_sq->allocate_gpu();
    a_layernorm6 = new Tensor({1, 256, 102});
    a_layernorm6->allocate_gpu();
    a_relu6 = new Tensor({1, 256, 102});
    a_relu6->allocate_gpu();
    a_pool6 = new Tensor({1, 256, 34});
    a_pool6->allocate_gpu();

    a_collapse = new Tensor({1, 8704});
    a_collapse->allocate_gpu();

    a_linear1 = new Tensor({1, 1024});
    a_linear1->allocate_gpu();
    a_relu7 = new Tensor({1, 1024});
    a_relu7->allocate_gpu();

    a_linear2 = new Tensor({1, 1024});
    a_linear2->allocate_gpu();
    a_relu8 = new Tensor({1, 1024});
    a_relu8->allocate_gpu();

    a_linear3 = new Tensor({1, 4});
    a_linear3->allocate_gpu();
  }
}

// Free all dynamically allocated variables
void finalize_classifier() {
  if (mpi_rank == 0) {
    delete w_conv1;
    delete b_conv1;
    delete w_conv2;
    delete b_conv2;
    delete w_conv3;
    delete b_conv3;
    delete w_conv4;
    delete b_conv4;
    delete w_conv5;
    delete b_conv5;
    delete w_conv6;
    delete b_conv6;
    delete w_fc1;
    delete b_fc1;
    delete w_fc2;
    delete b_fc2;
    delete w_fc3;
    delete b_fc3;
    delete gamma_conv1;
    delete gamma_conv6;
    delete beta_conv1;
    delete beta_conv6;
    delete a_conv1;
    delete a_layernorm1;
    delete a_relu1;
    delete a_pool1;
    delete a_conv2;
    delete a_relu2;
    delete a_pool2;
    delete a_conv3;
    delete a_relu3;
    delete a_conv4;
    delete a_relu4;
    delete a_conv5;
    delete a_relu5;
    delete a_conv6;
    delete a_layernorm6;
    delete a_relu6;
    delete a_pool6;
    delete a_collapse;
    delete a_linear1;
    delete a_relu7;
    delete a_linear2;
    delete a_relu8;
    delete a_linear3;
  }
}
