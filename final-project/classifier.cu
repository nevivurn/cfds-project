#include "hip/hip_runtime.h"
#include <math.h>
#include <mpi.h>
#include <cassert>

#include "classifier.h"
#include "util.h"

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

static int mpi_rank;

// Multi-dimensional matrix containing fp32 elements
struct Tensor {
  Tensor(std::vector<int> shape_);
  Tensor(std::vector<int> shape_, float *buf_);
  ~Tensor();
  __host__ __device__ int num_elem() const;
  void fill_zeros();

  void allocate_gpu();
  void copy_to_gpu(hipStream_t stream = 0);
  void copy_to_cpu(hipStream_t stream = 0);

  float *buf = nullptr;
  float *gbuf = nullptr;
  int ndim = 0;
  int shape[4];
  int datashape[4];
};

Tensor::Tensor(std::vector<int> shape_) {
  ndim = shape_.size();
  for (int i = 0; i < ndim; ++i) { shape[i] = datashape[i] = shape_[i]; }
  int N_ = num_elem();
  buf = (float *) calloc(N_, sizeof(float));
}

Tensor::Tensor(std::vector<int> shape_, float *buf_) {
  ndim = shape_.size();
  for (int i = 0; i < ndim; ++i) { shape[i] = datashape[i] = shape_[i]; }
  int N_ = num_elem();
  buf = (float *) calloc(N_, sizeof(float));
  for (int n = 0; n < N_; ++n) { buf[n] = buf_[n]; }
}

Tensor::~Tensor() {
  // TODO(nevi): properly free memory
  //if (buf != nullptr) free(buf);
  //if (gbuf != nullptr) hipFree(gbuf);
}

int Tensor::num_elem() const {
  int sz = 1;
  for (int i = 0; i < ndim; ++i) { sz *= shape[i]; }
  return sz;
}

void Tensor::fill_zeros() {
  int N_ = num_elem();
  for (int n = 0; n < N_; ++n) { buf[n] = 0.0; }
}

void Tensor::allocate_gpu() {
  CHECK_CUDA(hipMalloc(&gbuf, num_elem() * sizeof(float)));
}

void Tensor::copy_to_gpu(hipStream_t stream) {
  if (gbuf == nullptr)
    allocate_gpu();
  CHECK_CUDA(hipMemcpyAsync(gbuf, buf, num_elem() * sizeof(float), hipMemcpyHostToDevice, stream));
}

void Tensor::copy_to_cpu(hipStream_t stream) {
  CHECK_CUDA(hipMemcpyAsync(buf, gbuf, num_elem() * sizeof(float), hipMemcpyDeviceToHost, stream));
}

void print_tensor(Tensor *t, int n) {
  t->copy_to_cpu();
  CHECK_CUDA(hipStreamSynchronize(0));

  int n1 = t->shape[0];
  printf("size: %d %d %d\n", t->shape[0], t->shape[1], t->shape[2]);
  for (int i = 0; i < t->num_elem() / n1; ++i) {
    printf("%.4f ", t->buf[n * t->num_elem() / n1 + i]);
  }
}

// Parameters
Tensor *w_conv1, *w_conv2, *w_conv3, *w_conv4, *w_conv5, *w_conv6, *b_conv1,
    *b_conv2, *b_conv3, *b_conv4, *b_conv5, *b_conv6, *w_fc1, *w_fc2, *w_fc3,
    *b_fc1, *b_fc2, *b_fc3, *gamma_conv1, *beta_conv1, *gamma_conv6, *beta_conv6;

// Activations
Tensor *a_conv1, *a_layernorm1, *a_relu1, *a_pool1;
Tensor *a_conv1_sum, *a_conv1_sum_sq; // for layernorm
Tensor *a_conv2, *a_relu2, *a_pool2;
Tensor *a_conv3, *a_relu3;
Tensor *a_conv4, *a_relu4;
Tensor *a_conv5, *a_relu5;
Tensor *a_conv6, *a_layernorm6, *a_relu6, *a_pool6;
Tensor *a_conv6_sum, *a_conv6_sum_sq; // for layernorm
Tensor *a_collapse;
Tensor *a_linear1, *a_relu7;
Tensor *a_linear2, *a_relu8;
Tensor *a_linear3;

// Operations
void conv1d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride, int padding, int dilation, bool has_bias);
void relu(Tensor *input, Tensor *output);
void maxpool1d(Tensor *input, Tensor *output, int kernel_size, int stride);
void collapse(Tensor *input, Tensor *output);
void linear(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            bool has_bias);
void layernorm(Tensor *input, Tensor *gamma, Tensor *beta, Tensor *output);

// Cuda layers
__global__ void cuda_conv1d(Tensor input, Tensor weight, Tensor bias, Tensor output);
__global__ void cuda_relu(Tensor input, Tensor output);
__global__ void cuda_maxpool1d(Tensor input, Tensor output); // always 3x3
__global__ void cuda_collapse(Tensor input, Tensor output);
__global__ void cuda_linear(Tensor input, Tensor weight, Tensor bias, Tensor output);
__global__ void cuda_layernorm(Tensor input, Tensor gamma, Tensor beta, Tensor output,
                               Tensor sum, Tensor sum_sq);

// Cuda operations
__global__ void cuda_reduce_sum(Tensor input, Tensor output, int N);
__global__ void cuda_reduce_sum_sq(Tensor input, Tensor output, int N);

#define BATCH 512

// Parallelization method is totally up to you, but you should gather
// the output at rank 0
void classifier(float *input_, float *output_, int N) {
  assert(N % BATCH == 0);

  if (mpi_rank == 0) {
    for (int n = 0; n < N; n += BATCH) {  // N input sentences
      // Load BATCH sentences
      Tensor *input = new Tensor({BATCH, VOCAB_SIZE, MAX_LENGTH}, input_ + n * VOCAB_SIZE * MAX_LENGTH);

      // Conv block 1 : Conv1d + LayerNorm + ReLU + MaxPool1d
      input->copy_to_gpu();
      cuda_conv1d<<<dim3(1008, BATCH), 256>>>(*input, *w_conv1, *b_conv1, *a_conv1);
      cuda_reduce_sum<<<dim3(1008, BATCH), 256, 256 * sizeof(float)>>>(*a_conv1, *a_conv1_sum, 256);
      cuda_reduce_sum<<<dim3(BATCH, 1), 1024, 1024 * sizeof(float)>>>(*a_conv1_sum, *a_conv1_sum, 1008);
      cuda_reduce_sum_sq<<<dim3(1008, BATCH), 256, 256 * sizeof(float)>>>(*a_conv1, *a_conv1_sum_sq, 256);
      cuda_reduce_sum<<<dim3(BATCH, 1), 1024, 1024 * sizeof(float)>>>(*a_conv1_sum_sq, *a_conv1_sum_sq, 1008);
      cuda_layernorm<<<dim3(1008, BATCH), 256>>>(*a_conv1, *gamma_conv1, *beta_conv1, *a_layernorm1,
                                    *a_conv1_sum, *a_conv1_sum_sq);
      cuda_relu<<<BATCH*1008, 256>>>(*a_layernorm1, *a_relu1);
      cuda_maxpool1d<<<dim3(336, BATCH), 256>>>(*a_relu1, *a_pool1);

      // Conv block 2 : Conv1d + ReLU + MaxPool1d
      cuda_conv1d<<<dim3(330, BATCH), 256>>>(*a_pool1, *w_conv2, *b_conv2, *a_conv2);
      cuda_relu<<<BATCH*330, 256>>>(*a_conv2, *a_relu2);
      cuda_maxpool1d<<<dim3(110, BATCH), 256>>>(*a_relu2, *a_pool2);

      // Conv block 3 : Conv1d + ReLU
      cuda_conv1d<<<dim3(108, BATCH), 256>>>(*a_pool2, *w_conv3, *b_conv3, *a_conv3);
      cuda_relu<<<BATCH*108, 256>>>(*a_conv3, *a_relu3);

      // Conv block 4 : Conv1d + ReLU
      cuda_conv1d<<<dim3(106, BATCH), 256>>>(*a_relu3, *w_conv4, *b_conv4, *a_conv4);
      cuda_relu<<<BATCH*106, 256>>>(*a_conv4, *a_relu4);

      // Conv block 5 : Conv1d + ReLU
      cuda_conv1d<<<dim3(104, BATCH), 256>>>(*a_relu4, *w_conv5, *b_conv5, *a_conv5);
      cuda_relu<<<BATCH*104, 256>>>(*a_conv5, *a_relu5);

      // Conv block 6 : Conv1d + LayerNorm + ReLU + MaxPool1d
      cuda_conv1d<<<dim3(102, BATCH), 256>>>(*a_relu5, *w_conv6, *b_conv6, *a_conv6);
      cuda_reduce_sum<<<dim3(102, BATCH), 256, 256 * sizeof(float)>>>(*a_conv6, *a_conv6_sum, 256);
      cuda_reduce_sum<<<dim3(BATCH, 1), 128, 128 * sizeof(float)>>>(*a_conv6_sum, *a_conv6_sum, 102);
      cuda_reduce_sum_sq<<<dim3(102, BATCH), 256, 256 * sizeof(float)>>>(*a_conv6, *a_conv6_sum_sq, 256);
      cuda_reduce_sum<<<dim3(BATCH, 1), 128, 128 * sizeof(float)>>>(*a_conv6_sum_sq, *a_conv6_sum_sq, 102);
      cuda_layernorm<<<dim3(102, BATCH), 256>>>(*a_conv6, *gamma_conv6, *beta_conv6, *a_layernorm6,
                                    *a_conv6_sum, *a_conv6_sum_sq);
      cuda_relu<<<BATCH*102, 256>>>(*a_layernorm6, *a_relu6);
      cuda_maxpool1d<<<dim3(34, BATCH), 256>>>(*a_relu6, *a_pool6);

      // Collapse
      cuda_collapse<<<BATCH*68, 128>>>(*a_pool6, *a_collapse);

      // FC block 1 : Linear + ReLU
      cuda_linear<<<BATCH, 1024>>>(*a_collapse, *w_fc1, *b_fc1, *a_linear1);
      cuda_relu<<<BATCH, 1024>>>(*a_linear1, *a_relu7);

      // FC block 2 : Linear + ReLU
      cuda_linear<<<BATCH, 1024>>>(*a_relu7, *w_fc2, *b_fc2, *a_linear2);
      cuda_relu<<<BATCH, 1024>>>(*a_linear2, *a_relu8);

      // FC block 3 : Linear
      cuda_linear<<<BATCH, 4>>>(*a_relu8, *w_fc3, *b_fc3, *a_linear3);

      a_linear3->copy_to_cpu();
      CHECK_CUDA(hipStreamSynchronize(0));

      for (int i = 0; i < BATCH; i++) {
        float max_val = -1e99f;
        int max_idx = 0;
        for (int j = 0; j < a_linear3->shape[1]; j++) {
          if (a_linear3->buf[j + i * a_linear3->shape[1]] > max_val) {
            max_val = a_linear3->buf[j + i * a_linear3->shape[1]];
            max_idx = j;
          }
        }
        output_[n + i] = max_idx;
      }
    }  // end N input sentences loop
  }    // if mpi_rank == 0
}

__global__ void cuda_conv1d(Tensor input, Tensor weight, Tensor bias, Tensor output) {
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int IC = input.shape[1];
  int IL = input.shape[2];
  int IS = IC * IL;
  int OL = output.shape[2];
  int OS = output.shape[1] * output.shape[2];
  int KS = weight.shape[2];

  int oc = i / OL;
  int ol = i % OL;

  float val = bias.gbuf[oc];
  for (int ic = 0; ic < IC; ++ic)
    for (int ks = 0; ks < KS; ++ks)
      val += input.gbuf[ks + ol + ic * IL + j * IS] *
        weight.gbuf[ks + ic * KS + oc * IC * KS];
  output.gbuf[i + j * OS] = val;
}

void conv1d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride = 1, int padding = 0, int dilation = 1,
            bool has_bias = true) {
  int out_channels = weight->shape[0];
  int in_channels = weight->shape[1];
  int kernel_size = weight->shape[2];
  int input_length = input->shape[2];
  int output_length =
      (input->shape[2] + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

  for (int oc = 0; oc < out_channels; ++oc) {
    for (int ol = 0; ol < output_length; ++ol) {
      float val = 0.0f;
      int offset = ol;
      for (int ic = 0; ic < in_channels; ++ic) {
        for (int ks = 0; ks < kernel_size; ++ks) {
          val += weight->buf[oc * in_channels * kernel_size + ic * kernel_size + ks] *
                 input->buf[ic * input_length + ks + offset];
        }
      }
      if (has_bias) val += bias->buf[oc];
      output->buf[oc * output_length + ol] = val;
    }
  }
}


__global__ void cuda_relu(Tensor input, Tensor output) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (input.gbuf[i] > 0.0f)
    output.gbuf[i] = input.gbuf[i];
  else
    output.gbuf[i] = 0.0f;
}

void relu(Tensor *input, Tensor *output) {
  for (int i = 0; i < input->num_elem(); ++i) {
    if (input->buf[i] > 0.0f)
      output->buf[i] = input->buf[i];
    else
      output->buf[i] = 0.0f;
  }
}

__global__ void cuda_maxpool1d(Tensor input, Tensor output) {
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int IS = input.shape[1] * input.shape[2];
  int OS = output.shape[1] * output.shape[2];

  float mx = -1e99f;
  for (int ks = 0; ks < 3; ++ks) {
    float val = input.gbuf[ks + i * 3 + j * IS];
    if (val > mx) mx = val;
  }
  output.gbuf[i + j * OS] = mx;
}

void maxpool1d(Tensor *input, Tensor *output, int kernel_size, int stride) {
  int IL = input->shape[2];
  int OC = output->shape[1];
  int OL = output->shape[2];

  for (int oc = 0; oc < OC; ++oc) {
    for (int ol = 0; ol < OL; ++ol) {
      float mx = -1e99;
      for (int ks = 0; ks < kernel_size; ++ks) {
        float val = input->buf[oc * IL + ks + ol * stride];
        if (val > mx) mx = val;
      }
      output->buf[oc * OL + ol] = mx;
    }
  }
}

__global__ void cuda_collapse(Tensor input, Tensor output) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  output.gbuf[i] = input.gbuf[i];
}

void collapse(Tensor *input, Tensor *output) {
  for (int i = 0; i < input->num_elem(); ++i) {
    output->buf[i] = input->buf[i];
  }
}

__global__ void cuda_linear(Tensor input, Tensor weight, Tensor bias, Tensor output) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  int IC = input.shape[1];
  int OC = output.shape[1];

  float val = bias.gbuf[tid];
  for (int ic = 0; ic < IC; ++ic)
    val += input.gbuf[ic + bid * IC] * weight.gbuf[ic + tid * IC];
  output.gbuf[tid + bid * OC] = val;
}

void linear(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            bool has_bias) {
  int IC = input->shape[1];
  int OC = output->shape[1];

  for (int oc = 0; oc < OC; ++oc) {
    float val = 0.0;
    for (int ic = 0; ic < IC; ++ic) {
      val += input->buf[ic] * weight->buf[oc * IC + ic];
    }
    if (has_bias) val += bias->buf[oc];
    output->buf[oc] = val;
  }
}

__global__ void cuda_reduce_sum(Tensor input, Tensor output, int N) {
  extern __shared__ float sdata[];

  int IS = input.shape[1] * input.shape[2];
  int OS = output.shape[1];

  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int i = bid * blockDim.x + tid;

  if (tid < N)
    sdata[tid] = input.gbuf[i + j * IS];
  else
    sdata[tid] = 0.0f;
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    output.gbuf[bid + j * OS] = sdata[0];
}

__global__ void cuda_reduce_sum_sq(Tensor input, Tensor output, int N) {
  extern __shared__ float sdata[];

  int IS = input.shape[1] * input.shape[2];
  int OS = output.shape[1];

  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int i = bid * blockDim.x + tid;

  if (tid < N)
    sdata[tid] = input.gbuf[i + j * IS];
  else
    sdata[tid] = 0.0f;
  sdata[tid] *= sdata[tid];
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    output.gbuf[bid + j * OS] = sdata[0];
}

__global__ void cuda_layernorm(Tensor input, Tensor gamma, Tensor beta, Tensor output,
                               Tensor sum, Tensor sum_sq) {
  int IS = input.shape[1] * input.shape[2];

  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int off = i + j * IS;

  // E[X], E[X^2]
  float mean1 = sum.gbuf[j] / IS;
  float mean2 = sum_sq.gbuf[j] / IS;
  // V[X]
  float var = mean2 - mean1 * mean1;

  output.gbuf[off] =
    (input.gbuf[off] - mean1) / sqrtf(var + 1e-5) * gamma.gbuf[i] + beta.gbuf[i];
}

void layernorm(Tensor *input, Tensor *gamma, Tensor *beta, Tensor *output) {
  // E[X], E[X^2]
  float sum1 = 0.0f, sum2 = 0.0f;
  for (int i = 0; i < input->num_elem(); ++i) {
      sum1 += input->buf[i];
      sum2 += input->buf[i] * input->buf[i];
  }
  float mean1 = sum1 / (float)input->num_elem();
  float mean2 = sum2 / (float)input->num_elem();

  // V[X]
  float var = mean2 - mean1 * mean1;

  // Normalization
  for (int i = 0; i < input->num_elem(); ++i) {
    output->buf[i] = (input->buf[i] - mean1) / sqrtf(var + 1e-5) * gamma->buf[i] + beta->buf[i];
  }
}

// load the parameter binary file and store parameters into Tensors
// Only the first process (root, mpi_rank == 0) has the parameter
// You must broadcast it to the others
void initialize_classifier(float *parameter, int N) {
  CHECK_CUDA(hipSetDevice(0));

  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  if (mpi_rank == 0) {
    w_conv1 = new Tensor({256, 70, 7}, parameter + OFFSET0);
    w_conv1->copy_to_gpu();
    b_conv1 = new Tensor({256}, parameter + OFFSET1);
    b_conv1->copy_to_gpu();
    gamma_conv1 = new Tensor({256, 1008}, parameter + OFFSET2);
    gamma_conv1->copy_to_gpu();
    beta_conv1 = new Tensor({256, 1008}, parameter + OFFSET3);
    beta_conv1->copy_to_gpu();

    w_conv2 = new Tensor({256, 256, 7}, parameter + OFFSET4);
    w_conv2->copy_to_gpu();
    b_conv2 = new Tensor({256}, parameter + OFFSET5);
    b_conv2->copy_to_gpu();
    w_conv3 = new Tensor({256, 256, 3}, parameter + OFFSET6);
    w_conv3->copy_to_gpu();
    b_conv3 = new Tensor({256}, parameter + OFFSET7);
    b_conv3->copy_to_gpu();
    w_conv4 = new Tensor({256, 256, 3}, parameter + OFFSET8);
    w_conv4->copy_to_gpu();
    b_conv4 = new Tensor({256}, parameter + OFFSET9);
    b_conv4->copy_to_gpu();
    w_conv5 = new Tensor({256, 256, 3}, parameter + OFFSET10);
    w_conv5->copy_to_gpu();
    b_conv5 = new Tensor({256}, parameter + OFFSET11);
    b_conv5->copy_to_gpu();

    w_conv6 = new Tensor({256, 256, 3}, parameter + OFFSET12);
    w_conv6->copy_to_gpu();
    b_conv6 = new Tensor({256}, parameter + OFFSET13);
    b_conv6->copy_to_gpu();
    gamma_conv6 = new Tensor({256, 102}, parameter + OFFSET14);
    gamma_conv6->copy_to_gpu();
    beta_conv6 = new Tensor({256, 102}, parameter + OFFSET15);
    beta_conv6->copy_to_gpu();
    w_fc1 = new Tensor({1024, 8704}, parameter + OFFSET16);
    w_fc1->copy_to_gpu();
    b_fc1 = new Tensor({1024}, parameter + OFFSET17);
    b_fc1->copy_to_gpu();
    w_fc2 = new Tensor({1024, 1024}, parameter + OFFSET18);
    w_fc2->copy_to_gpu();
    b_fc2 = new Tensor({1024}, parameter + OFFSET19);
    b_fc2->copy_to_gpu();
    w_fc3 = new Tensor({4, 1024}, parameter + OFFSET20);
    w_fc3->copy_to_gpu();
    b_fc3 = new Tensor({4}, parameter + OFFSET21);
    b_fc3->copy_to_gpu();

    a_conv1 = new Tensor({BATCH, 256, 1008});
    a_conv1->allocate_gpu();
    a_conv1_sum = new Tensor({BATCH, 1024}); // rounded up
    a_conv1_sum->allocate_gpu();
    a_conv1_sum_sq = new Tensor({BATCH, 1024});
    a_conv1_sum_sq->allocate_gpu();
    a_layernorm1 = new Tensor({BATCH, 256, 1008});
    a_layernorm1->allocate_gpu();
    a_relu1 = new Tensor({BATCH, 256, 1008});
    a_relu1->allocate_gpu();
    a_pool1 = new Tensor({BATCH, 256, 336});
    a_pool1->allocate_gpu();

    a_conv2 = new Tensor({BATCH, 256, 330});
    a_conv2->allocate_gpu();
    a_relu2 = new Tensor({BATCH, 256, 330});
    a_relu2->allocate_gpu();
    a_pool2 = new Tensor({BATCH, 256, 110});
    a_pool2->allocate_gpu();

    a_conv3 = new Tensor({BATCH, 256, 108});
    a_conv3->allocate_gpu();
    a_relu3 = new Tensor({BATCH, 256, 108});
    a_relu3->allocate_gpu();

    a_conv4 = new Tensor({BATCH, 256, 106});
    a_conv4->allocate_gpu();
    a_relu4 = new Tensor({BATCH, 256, 106});
    a_relu4->allocate_gpu();

    a_conv5 = new Tensor({BATCH, 256, 104});
    a_conv5->allocate_gpu();
    a_relu5 = new Tensor({BATCH, 256, 104});
    a_relu5->allocate_gpu();

    a_conv6 = new Tensor({BATCH, 256, 102});
    a_conv6->allocate_gpu();
    a_conv6_sum = new Tensor({BATCH, 128}); // rounded up
    a_conv6_sum->allocate_gpu();
    a_conv6_sum_sq = new Tensor({BATCH, 128});
    a_conv6_sum_sq->allocate_gpu();
    a_layernorm6 = new Tensor({BATCH, 256, 102});
    a_layernorm6->allocate_gpu();
    a_relu6 = new Tensor({BATCH, 256, 102});
    a_relu6->allocate_gpu();
    a_pool6 = new Tensor({BATCH, 256, 34});
    a_pool6->allocate_gpu();

    a_collapse = new Tensor({BATCH, 8704});
    a_collapse->allocate_gpu();

    a_linear1 = new Tensor({BATCH, 1024});
    a_linear1->allocate_gpu();
    a_relu7 = new Tensor({BATCH, 1024});
    a_relu7->allocate_gpu();

    a_linear2 = new Tensor({BATCH, 1024});
    a_linear2->allocate_gpu();
    a_relu8 = new Tensor({BATCH, 1024});
    a_relu8->allocate_gpu();

    a_linear3 = new Tensor({BATCH, 4});
    a_linear3->allocate_gpu();
  }
}

// Free all dynamically allocated variables
void finalize_classifier() {
  if (mpi_rank == 0) {
    delete w_conv1;
    delete b_conv1;
    delete w_conv2;
    delete b_conv2;
    delete w_conv3;
    delete b_conv3;
    delete w_conv4;
    delete b_conv4;
    delete w_conv5;
    delete b_conv5;
    delete w_conv6;
    delete b_conv6;
    delete w_fc1;
    delete b_fc1;
    delete w_fc2;
    delete b_fc2;
    delete w_fc3;
    delete b_fc3;
    delete gamma_conv1;
    delete gamma_conv6;
    delete beta_conv1;
    delete beta_conv6;
    delete a_conv1;
    delete a_layernorm1;
    delete a_relu1;
    delete a_pool1;
    delete a_conv2;
    delete a_relu2;
    delete a_pool2;
    delete a_conv3;
    delete a_relu3;
    delete a_conv4;
    delete a_relu4;
    delete a_conv5;
    delete a_relu5;
    delete a_conv6;
    delete a_layernorm6;
    delete a_relu6;
    delete a_pool6;
    delete a_collapse;
    delete a_linear1;
    delete a_relu7;
    delete a_linear2;
    delete a_relu8;
    delete a_linear3;
  }
}
